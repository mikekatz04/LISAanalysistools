#include "hip/hip_runtime.h"
#include "stdio.h"
#include "global.hpp"
#include "Detector.hpp"
#include <iostream>
#include <stdexcept>
#include <string>
#include <sstream>

CUDA_DEVICE
int Orbits::get_window(double t)
{
    int out = int(t / dt);
    if ((out < 0) || (out >= N))
        return -1;
    else
        return out;
}

CUDA_DEVICE
int Orbits::get_link_ind(int link)
{
    if (link == 12)
        return 0;
    else if (link == 23)
        return 1;
    else if (link == 31)
        return 2;
    else if (link == 13)
        return 3;
    else if (link == 32)
        return 4;
    else if (link == 21)
        return 5;
    else
#ifdef __HIPCC__
        printf("BAD link ind. Must be 12, 23, 31, 13, 32, 21.");
#else
        throw std::invalid_argument("Bad link ind. Must be 12, 23, 31, 13, 32, 21.");
#endif // __HIPCC__
    return -1;
}

CUDA_DEVICE
int Orbits::get_sc_ind(int sc)
{
    if (sc == 1)
        return 0;
    else if (sc == 2)
        return 1;
    else if (sc == 3)
        return 2;
    else
    {
#ifdef __HIPCC__
        printf("BAD sc ind. Must be 1,2,3. %d\n", sc);
#else
        std::ostringstream oss;
        oss << "Bad sc ind. Must be 1,2,3. Input sc is " << sc << " " << std::endl;
        std::string var = oss.str();
        throw std::invalid_argument(var);
#endif // __HIPCC__
    }
    return 0;
}

CUDA_DEVICE
double Orbits::interpolate(double t, double *in_arr, int window, int major_ndim, int major_ind, int ndim, int pos)
{
    double up = in_arr[((window + 1) * major_ndim + major_ind) * ndim + pos]; // down_ind * ndim + pos];
    double down = in_arr[(window * major_ndim + major_ind) * ndim + pos];

    // m *(x - x0) + y0
    double fin = ((up - down) / dt) * (t - (dt * window)) + down;
    // if ((ndim == 1))
    //     printf("%d %e %e %e %e \n", window, fin, down, up, (t - (dt * window)));

    return fin;
}

CUDA_DEVICE
void Orbits::get_normal_unit_vec_ptr(Vec *vec, double t, int link)
{
    Vec _tmp = get_normal_unit_vec(t, link);
    vec->x = _tmp.x;
    vec->y = _tmp.y;
    vec->z = _tmp.z;
}

CUDA_DEVICE
Vec Orbits::get_normal_unit_vec(double t, int link)
{
    int window = get_window(t);
    if (window == -1)
    {
        // out of bounds
        return Vec(0.0, 0.0, 0.0);
    }

    int link_ind = get_link_ind(link);

    int up_ind = (window + 1) * nlinks + link_ind;
    int down_ind = window * nlinks + link_ind;

    // x (pos = 0) ndim = 3
    double x_out = interpolate(t, n_arr, window, nlinks, link_ind, 3, 0);
    // y (pos = 1)
    double y_out = interpolate(t, n_arr, window, nlinks, link_ind, 3, 1);
    // z (pos = 2)
    double z_out = interpolate(t, n_arr, window, nlinks, link_ind, 3, 2);

    return Vec(x_out, y_out, z_out);
}

CUDA_DEVICE
double Orbits::get_light_travel_time(double t, int link)
{
    int window = get_window(t);
    if (window == -1)
    {
        // out of bounds
        return 0.0;
    }

    int link_ind = get_link_ind(link);
    if ((link_ind < 0) || (link_ind >= 6))
        printf("BAD %d\n", link_ind);
    int up_ind = (window + 1) * (nlinks + link_ind);
    int down_ind = window * (nlinks + link_ind);

    // x (pos = 0), ndim = 1
    double ltt_out = interpolate(t, ltt_arr, window, nlinks, link_ind, 1, 0);

    return ltt_out;
}

CUDA_DEVICE
Vec Orbits::get_pos(double t, int sc)
{
    int window = get_window(t);
    if (window == -1)
    {
        // out of bounds
        return Vec(0.0, 0.0, 0.0);
    }

    int sc_ind = get_sc_ind(sc);

    // x (pos = 0), ndim = 3
    double x_out = interpolate(t, x_arr, window, nspacecraft, sc_ind, 3, 0);
    // y (pos = 1), ndim = 3
    double y_out = interpolate(t, x_arr, window, nspacecraft, sc_ind, 3, 1);
    // z (pos = 2), ndim = 3
    double z_out = interpolate(t, x_arr, window, nspacecraft, sc_ind, 3, 2);
    return Vec(x_out, y_out, z_out);
}

CUDA_DEVICE
void Orbits::get_pos_ptr(Vec *vec, double t, int sc)
{
    Vec _tmp = get_pos(t, sc);
    vec->x = _tmp.x;
    vec->y = _tmp.y;
    vec->z = _tmp.z;
}

#define NUM_THREADS 64


CUDA_KERNEL
void get_light_travel_time_kernel(double *ltt, double *t, int *link, int num, Orbits &orbits)
{
    int start, end, increment;
#ifdef __HIPCC__
    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num;
    increment = gridDim.x * blockDim.x;
#else  // __HIPCC__
    start = 0;
    end = num;
    increment = 1;
#endif // __HIPCC__

    for (int i = start; i < end; i += increment)
    {
        ltt[i] = orbits.get_light_travel_time(t[i], link[i]);
    }
}

void Orbits::get_light_travel_time_arr(double *ltt, double *t, int *link, int num)
{
#ifdef __HIPCC__
    int num_blocks = std::ceil((num + NUM_THREADS - 1) / NUM_THREADS);

    // copy self to GPU
    Orbits *orbits_gpu;
    gpuErrchk(hipMalloc(&orbits_gpu, sizeof(Orbits)));
    gpuErrchk(hipMemcpy(orbits_gpu, this, sizeof(Orbits), hipMemcpyHostToDevice));

    get_light_travel_time_kernel<<<num_blocks, NUM_THREADS>>>(ltt, t, link, num, *orbits_gpu);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipFree(orbits_gpu));

#else // __HIPCC__

    get_light_travel_time_kernel(ltt, t, link, num, *this);

#endif // __HIPCC__
}


CUDA_KERNEL
void get_pos_kernel(double *pos_x, double *pos_y, double *pos_z, double *t, int *sc, int num, Orbits &orbits)
{
    int start, end, increment;
#ifdef __HIPCC__
    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num;
    increment = gridDim.x * blockDim.x;
#else  // __HIPCC__
    start = 0;
    end = num;
    increment = 1;
#endif // __HIPCC__
    Vec _tmp(0.0, 0.0, 0.0);

    for (int i = start; i < end; i += increment)
    {
        _tmp = orbits.get_pos(t[i], sc[i]);
        pos_x[i] = _tmp.x;
        pos_y[i] = _tmp.y;
        pos_z[i] = _tmp.z;
    }
}

void Orbits::get_pos_arr(double *pos_x, double *pos_y, double *pos_z, double *t, int *sc, int num)
{
#ifdef __HIPCC__
    int num_blocks = std::ceil((num + NUM_THREADS - 1) / NUM_THREADS);

    // copy self to GPU
    Orbits *orbits_gpu;
    gpuErrchk(hipMalloc(&orbits_gpu, sizeof(Orbits)));
    gpuErrchk(hipMemcpy(orbits_gpu, this, sizeof(Orbits), hipMemcpyHostToDevice));

    get_pos_kernel<<<num_blocks, NUM_THREADS>>>(pos_x, pos_y, pos_z, t, sc, num, *orbits_gpu);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipFree(orbits_gpu));

#else // __HIPCC__

    get_pos_kernel(pos_x, pos_y, pos_z, t, sc, num, *this);

#endif // __HIPCC__
}


CUDA_KERNEL
void get_normal_unit_vec_kernel(double *normal_unit_vec_x, double *normal_unit_vec_y, double *normal_unit_vec_z, double *t, int *link, int num, Orbits &orbits)
{
    int start, end, increment;
#ifdef __HIPCC__
    start = blockIdx.x * blockDim.x + threadIdx.x;
    end = num;
    increment = gridDim.x * blockDim.x;
#else  // __HIPCC__
    start = 0;
    end = num;
    increment = 1;
#endif // __HIPCC__
    Vec _tmp(0.0, 0.0, 0.0);

    for (int i = start; i < end; i += increment)
    {
        _tmp = orbits.get_normal_unit_vec(t[i], link[i]);
        normal_unit_vec_x[i] = _tmp.x;
        normal_unit_vec_y[i] = _tmp.y;
        normal_unit_vec_z[i] = _tmp.z;
    }
}

void Orbits::get_normal_unit_vec_arr(double *normal_unit_vec_x, double *normal_unit_vec_y, double *normal_unit_vec_z, double *t, int *link, int num)
{
#ifdef __HIPCC__
    int num_blocks = std::ceil((num + NUM_THREADS - 1) / NUM_THREADS);

    // copy self to GPU
    Orbits *orbits_gpu;
    gpuErrchk(hipMalloc(&orbits_gpu, sizeof(Orbits)));
    gpuErrchk(hipMemcpy(orbits_gpu, this, sizeof(Orbits), hipMemcpyHostToDevice));

    get_normal_unit_vec_kernel<<<num_blocks, NUM_THREADS>>>(normal_unit_vec_x, normal_unit_vec_y, normal_unit_vec_z, t, link, num, *orbits_gpu);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipFree(orbits_gpu));

#else // __HIPCC__

    get_normal_unit_vec_kernel(normal_unit_vec_x, normal_unit_vec_y, normal_unit_vec_z, t, link, num, *this);

#endif // __HIPCC__
}

